#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZE 100 //N can not be larger than 256


void add_matrix_cpu(int *a, int *b, int *c, int N){
	int i, j index;
	for(i=0; i<N; i++){
		for(j = 0; j<N; j++){
			index = i*N+j;
			c[index] = a[index] + b[index];
		}
	}
}

__global__ void add_matrix_gpu(int *a, int *b, int *c, int N){
	int col = blockId.x * blockDim.x + threadId.x;
	int row = blockId.y * blockDim.y + threadID.y;
	int index = row * N + col;
	if( row < N && col < N) {
		c[index] = a[index] + b[index];
	}
}
