#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
//Code writen by Alan Fleming

void add_matrix_cpu(int *a, int *b, int *c, int N){
	int i, j, index;
	for( i = 0; i<N; i++){
		for( j = 0; j<N; j++){
			index = i*N+j;
			c[index] = a[index] + b[index];
		}
	}
}

__global__ void add_matrix_gpu(int *a, int *b, int *c, int N){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * N + col;
	if( row < N && col < N) {
		c[index] = a[index] + b[index];
	}
}

void printMatrix(int *m, int N){
	for( int i = 0; i < N; i++){
		for( int j = 0; j < N; j++){
			printf("%d ", m[i * N + j]);
		}
		printf("\n");
	}
}


int verifyMatrix(int *a, int *b, int N){
	for( int i = 0; i < N; i++){
		for( int j = 0; j < N; j++){
			if(a[i * N + j] != b[i * N + j]){
				printf("TEST FAILED\n");
				return 1;
			}
		}
	}
	printf("TEST PASSED\n");
	return 0;
}

int main(int argc, char *argv[]){

	//assign matrix and block size
	const int MATRIXSIZE = argv[1];
	const int BLOCKSIZE = argv[2];

	//allocate system memory for array
	int *a = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE );	//first matrix
	int *b = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE ); //second matrix
	int *c = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE ); //result from CPU
	int *d = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE ); //result from gpu

	//initialize a and b for addition
	int init = 1325;
	for( int i = 0; i < MATRIXSIZE; i++){
		for( int j = 0; j < MATRIXSIZE; j++){
			init = 3125 * init % 65536;
			a[ i * MATRIXSIZE + j ] = (init - 32768)/6553;
			b[ i * MATRIXSIZE + j ] = init % 1000;
		}
	}

	//print initial matrix a and b
	printf("a \n --------------------- \n");
	printMatrix(a, MATRIXSIZE);

	printf("b \n --------------------- \n");
	printMatrix(b, MATRIXSIZE);

	//add matrix using cpu
	add_matrix_cpu(a, b, c, MATRIXSIZE);
	
	//print the result
	printf("c \n --------------------- \n");
	printMatrix(c, MATRIXSIZE);

	//allocate memory on device
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **)(&dev_a),MATRIXSIZE * MATRIXSIZE * sizeof(int));
	hipMalloc((void **)(&dev_b),MATRIXSIZE * MATRIXSIZE * sizeof(int));
	hipMalloc((void **)(&dev_c),MATRIXSIZE * MATRIXSIZE * sizeof(int));

	//copy memory to device
	hipMemcpy(dev_a,a, MATRIXSIZE * MATRIXSIZE * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b, MATRIXSIZE * MATRIXSIZE * sizeof(int),hipMemcpyHostToDevice);
	
	//calculate gridWidth
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);

	int gridWidth = ceil((MATRIXSIZE-1)/double(dimBlock.x));

	//define dimGrid
	dim3 dimGrid(gridWidth, gridWidth,1);

	//add matrix using gpu
	add_matrix_gpu<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, MATRIXSIZE);

	//copy memory from device
	hipMemcpy(d,dev_c, MATRIXSIZE * MATRIXSIZE * sizeof(int),hipMemcpyDeviceToHost);

	//print the result
	printf("d \n --------------------- \n");
	printMatrix(d, MATRIXSIZE);

	//verify the results
	verifyMatrix(c, d, MATRIXSIZE);

	//free memory
	free(a);
        free(b);
        free(c);
        free(d); 
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
	//exit program
	return 0;
}
