#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define MATRIXSIZE 8 //N can not be larger than 256
#define BLOCKSIZE 4

void mul_matrix_cpu(int *M, int *N, int *P, int width){
	for( int i = 0; i<width; i++){
		for( int j = 0; j<width; j++){
			int sum = 0;
			for (int k = 0; k < width; k++){
				sum += M[i * width + k] * N[i * width + j];
			}
			p[i * width + j] = sum;
		}
	}
}

__global__ void mul_matrix_gpu(int *M, int *N, int *P, int width){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if( row < width && col < width) {
		int pValue = 0;
		for( k = 0; k < width; k++){
			pValue =+ M[row * width + k] * N[k * width + col];
		}
		P[row * width + col] = pValue;
	}
}

void printMatrix(int *m, int N){
	for( int i = 0; i < N; i++){
		for( int j = 0; j < N; j++){
			printf("%d ", m[i * N + j]);
		}
		printf("\n");
	}
}


int verifyMatrix(int *a, int *b, int N){
	for( int i = 0; i < N; i++){
		for( int j = 0; j < N; j++){
			if(a[i * N + j] != b[i * N + j]){
				printf("TEST FAILED\n");
				return 1;
			}
		}
	}
	printf("TEST PASSED\n");
	return 0;
}

int main(){

	//allocate system memory for array
	int *a = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE );	//first matrix
	int *b = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE ); //second matrix
	int *c = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE ); //result from CPU
	int *d = (int *)malloc(sizeof(int) * MATRIXSIZE * MATRIXSIZE ); //result from gpu

	//initialize a and b for addition
	int init = 1325;
	for( int i = 0; i < MATRIXSIZE; i++){
		for( int j = 0; j < MATRIXSIZE; j++){
			init = 3125 * init % 65536;
			a[ i * MATRIXSIZE + j ] = (init - 32768)/6553;
			b[ i * MATRIXSIZE + j ] = init % 1000;
		}
	}

	//print initial matrix a and b
	printf("a \n --------------------- \n");
	printMatrix(a, MATRIXSIZE);

	printf("b \n --------------------- \n");
	printMatrix(b, MATRIXSIZE);

	//multiply matrix using cpu
	mul_matrix_cpu(a, b, c, MATRIXSIZE);
	
	//print the result
	printf("c \n --------------------- \n");
	printMatrix(c, MATRIXSIZE);

	//allocate memory on device
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **)(&dev_a),MATRIXSIZE * MATRIXSIZE * sizeof(int));
	hipMalloc((void **)(&dev_b),MATRIXSIZE * MATRIXSIZE * sizeof(int));
	hipMalloc((void **)(&dev_c),MATRIXSIZE * MATRIXSIZE * sizeof(int));

	//copy memory to device
	hipMemcpy(dev_a,a, MATRIXSIZE * MATRIXSIZE * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b, MATRIXSIZE * MATRIXSIZE * sizeof(int),hipMemcpyHostToDevice);
	
	//calculate gridWidth
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);

	int gridWidth = ceil((MATRIXSIZE-1)/double(dimBlock.x));

	//define dimGrid
	dim3 dimGrid(gridWidth, gridWidth,1);

	//multiply matrix using gpu
	mul_matrix_gpu<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, MATRIXSIZE);

	//copy memory from device
	hipMemcpy(d,dev_c, MATRIXSIZE * MATRIXSIZE * sizeof(int),hipMemcpyDeviceToHost);

	//print the result
	printf("d \n --------------------- \n");
	printMatrix(d, MATRIXSIZE);

	//verify the results
	verifyMatrix(c, d, MATRIXSIZE);

	//free memory
	free(a);
        free(b);
        free(c);
        free(d); 
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
	//exit program
	return 0;
}
