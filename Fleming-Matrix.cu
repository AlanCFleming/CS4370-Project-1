#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZE 100 //N can not be larger than 256


void add_matrix_cpu(int *a, int *b, int *c, int N){
	int i, j index;
	for(i=0; i<N; i++){
		for(j = 0; j<N; j++){
			index = i*N+j;
			c[index] = a[index] + b[index];
		}
	}
}
